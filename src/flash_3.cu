#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include "utils.h"
#include "flash.h"
using namespace nvcuda;

__global__
void forward_kernel_3(mykernelParamType2 param) {
    int tx = threadIdx.x;
    int warp_id = tx / 32;
    int bx = blockIdx.x; int by = blockIdx.y; int bz = blockIdx.z; 

    int kv_offset = (bz * gridDim.y * param.N * param.d) + (by * param.N * param.d);
    int q_offset = kv_offset + bx * param.d * param.Br;

    half*  Q = param.Q + q_offset;
    half*  K = param.K + kv_offset;
    half*  V = param.V + kv_offset;
    float* O = param.O + q_offset;

    float row_m_prev = -INFINITY;
    float row_l_prev = 0;
    const int load_Q_num  = param.Br * param.d / blockDim.x;  // 每个线程从全局内存搬运Q到共享内存的数据量
    const int load_KV_num = param.Bc * param.d / blockDim.x;  // 每个线程从全局内存搬运KV到共享内存的数据量

    extern __shared__ half sram[];
    const int tile_size = param.Bc * param.d;
    half* Qj     = sram;
    half* Kj     = &sram[param.Br * param.d];
    half* Vj     = &sram[param.Br * param.d + tile_size];
    half* S_half = &sram[param.Br * param.d + 2 * tile_size];

    __shared__ float S_mem[Br2 * HEAD_EMBD];
    float* S = S_mem;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> d_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;;
    
    float scores[8];

    #pragma unroll
    for (int i = 0; i < load_Q_num; i++){
        Qj[tx * load_Q_num + i] = Q[tx * load_Q_num + i];
    }
    __syncthreads();

    for (int i = 0; i < param.Tc; i++){
        
        #pragma unroll
        for (int j = 0; j < load_KV_num; j++){
            Kj[tx * load_KV_num + j] = K[tx * load_KV_num + j];
            Vj[tx * load_KV_num + j] = V[tx * load_KV_num + j];
        }
        
        K += tile_size;
        V += tile_size;

        __syncthreads();
        
        // S = QK^T
        wmma::fill_fragment(c_frag, 0.0f);

        #pragma unroll
        for (int x = 0; x < param.d / 16; x++){
            const half* aOffsetPtr = Qj + (warp_id / (Bc2/16)) * param.d * 16 + 16 * x;
            const half* bOffsetPtr = Kj + (warp_id % (Bc2/16)) * param.d * 16 + 16 * x;

            load_matrix_sync(a_frag, aOffsetPtr, param.d);
            load_matrix_sync(b_frag, bOffsetPtr, param.d);

            mma_sync(c_frag, a_frag, b_frag, c_frag);
        }

        float* cOffsetPtr = S + (warp_id / (Bc2/16)) * param.Bc * 16 + (warp_id % (Bc2/16)) * 16;

        store_matrix_sync(cOffsetPtr, c_frag, param.Bc, wmma::mem_row_major);

        __syncthreads();

        // row_m = rowmax(S)
        float row_m = -INFINITY;
        for (int x = 0; x < 8; x++){
            scores[x] = S[tx * 8 + x] * param.softmax_scale;
            if (scores[x] > row_m)    row_m = scores[x];
        }

        #pragma unroll
        for (int x = Bc2/8-1; x >= 1; x /= 2){
            float row_m_other = __shfl_xor_sync(0xffffffff, row_m, x, Bc2/8);
            row_m = fmaxf(row_m, row_m_other);
        }

        // P = exp(S - row_m), row_l = rowsum(P)
        float row_l = 0;
        #pragma unroll
        for (int x = 0; x < 8; x++) {
            scores[x] = __expf(scores[x] - row_m);
            row_l += scores[x];
        }

        #pragma unroll
        for (int x = Bc2/8-1; x >= 1; x /= 2){
            float row_l_other = __shfl_xor_sync(0xffffffff, row_l, x, Bc2/8);
            row_l += row_l_other;
        }

        #pragma unroll
        for (int x = 0; x < 8; x++){
            S_half[tx * 8 + x] = __float2half(scores[x]);
        }
        __syncthreads();    

        float row_m_new = row_m_prev > row_m ? row_m_prev : row_m;
        float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

        // S = S * V
        #pragma unroll
        for (int x = 0; x < param.d / Bc2; x++){
            wmma::fill_fragment(c_frag, 0.0f);
            #pragma unroll
            for(int y = 0; y < param.Bc / 16; y++){
                const half* aOffsetPtr = S_half + (warp_id / (Bc2/16)) * param.Bc * 16 + 16 * y;
                const half* dOffsetPtr = Vj + (warp_id % (Bc2/16)) * param.d / (Bc2/16) + 16 * x + y * param.d * 16;

                load_matrix_sync(a_frag, aOffsetPtr, param.Bc);
                load_matrix_sync(d_frag, dOffsetPtr, param.d);
                
                mma_sync(c_frag, a_frag, d_frag, c_frag);
            }

            float* cOffsetPtr = S + (warp_id / (Bc2/16)) * param.d * 16 + (warp_id % (Bc2/16)) * param.d / (Bc2/16) + x * 16;

            store_matrix_sync(cOffsetPtr, c_frag, param.d, wmma::mem_row_major);
        }
    
        __syncthreads();

        #pragma unroll
        for (int x = 0; x < param.d * param.Br / blockDim.x; x++){
            O[tx * param.d * param.Br / blockDim.x + x] = (1 / row_l_new) * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[tx * param.d * param.Br / blockDim.x + x]) \
                + (__expf(row_m - row_m_new) * S[tx * param.d * param.Br / blockDim.x + x]));
        }

        __syncthreads();

        row_l_prev = row_l_new;
        row_m_prev = row_m_new;
    }
}