#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cstdint>
#include "utils.h"
#include "flash.h"
using namespace nvcuda;

__global__
void forward_kernel_4(mykernelParamType2 param) {
    int tx = threadIdx.x;
    int warp_id = tx / 32; int lane_id = tx % 32;
    int bx = blockIdx.x; int by = blockIdx.y; int bz = blockIdx.z; 

    int kv_offset = (bz * gridDim.y * param.N * param.d) + (by * param.N * param.d);
    int q_offset = kv_offset + bx * param.d * param.Br;

    half*  Q = param.Q + q_offset;
    half*  K = param.K + kv_offset;
    half*  V = param.V + kv_offset;
    float* O = param.O + q_offset;

    extern __shared__ half sram[];
    int tile_size = param.Bc * param.d;
    half* Qj     = sram;
    half* Kj     = &sram[param.Br * param.d];
    half* Vj     = &sram[param.Br * param.d + tile_size];

    float row_l_prev1 = 0;
    float row_l_prev2 = 0;
    float row_m_prev1 = -INFINITY;
    float row_m_prev2 = -INFINITY;

    const int load_Q_num  = param.Br * param.d / blockDim.x;
    const int load_KV_num = param.Bc * param.d / blockDim.x;
    
    uint32_t a_frag[4];
    uint32_t b_frag[4];
    float    c_frag[Bc4/16][8];
    uint32_t d_frag[Bc4/16][4];

    #pragma unroll
    for (int i = 0; i < load_Q_num; i++){
        Qj[tx * load_Q_num + i] = Q[tx * load_Q_num + i];
    }

    __syncthreads();

    for (int i = 0; i < param.Tc; i++){
        
        #pragma unroll
        for (int j = 0; j < load_KV_num; j++){
            Kj[tx * load_KV_num + j] = K[tx * load_KV_num + j];
            Vj[tx * load_KV_num + j] = V[tx * load_KV_num + j];
        }
        
        K += tile_size;
        V += tile_size;

        __syncthreads();
        
        // S = QK^T
        memset(c_frag, 0, sizeof(c_frag));
        #pragma unroll
        for (int x = 0; x < param.d / 16; x++){
            uint32_t aOffsetPtr = __cvta_generic_to_shared(&Qj[warp_id*param.d*16+x*16+(lane_id%16)*param.d+(lane_id/16)*8]);
            LDMATRIX_X4(a_frag[0], a_frag[1], a_frag[2], a_frag[3], aOffsetPtr);
            #pragma unroll
            for (int y = 0; y < param.Bc / 16; y++){
                uint32_t bOffsetPtr = __cvta_generic_to_shared(&Kj[y*param.d*16+x*16+(lane_id%16)*param.d+(lane_id/16)*8]);
                LDMATRIX_X4(b_frag[0], b_frag[2], b_frag[1], b_frag[3], bOffsetPtr);
                __syncwarp();

                HMMA16816F32(c_frag[y][0], c_frag[y][1], c_frag[y][4], c_frag[y][5], \
                             a_frag[0], a_frag[1], a_frag[2], a_frag[3], \
                             b_frag[0], b_frag[1], \
                             c_frag[y][0], c_frag[y][1], c_frag[y][4], c_frag[y][5]);

                HMMA16816F32(c_frag[y][2], c_frag[y][3], c_frag[y][6], c_frag[y][7], \
                             a_frag[0], a_frag[1], a_frag[2], a_frag[3], \
                             b_frag[2], b_frag[3], \
                             c_frag[y][2], c_frag[y][3], c_frag[y][6], c_frag[y][7]);
            }
        }

        __syncthreads();

        float row_m1 = -INFINITY;
        float row_m2 = -INFINITY;
        #pragma unroll
        for (int x = 0; x < param.Bc / 16; x++){
            #pragma unroll
            for (int y = 0; y < 4; y++){
                c_frag[x][y]     *= param.softmax_scale;
                c_frag[x][y + 4] *= param.softmax_scale;
                if (c_frag[x][y]     > row_m1)    row_m1 = c_frag[x][y]    ;
                if (c_frag[x][y + 4] > row_m2)    row_m2 = c_frag[x][y + 4];
            }
        }

        #pragma unroll
        for (int x = 3; x >= 1; x /= 2){
            float row_m_other = __shfl_xor_sync(0xffffffff, row_m1, x, 4);
            row_m1 = fmaxf(row_m1, row_m_other);
            row_m_other = __shfl_xor_sync(0xffffffff, row_m2, x, 4);
            row_m2 = fmaxf(row_m2, row_m_other);
        }

        float row_l1 = 0;
        float row_l2 = 0;
        #pragma unroll
        for (int x = 0; x < param.Bc / 16; x++) {
            #pragma unroll
            for (int y = 0; y < 4; y++){
                c_frag[x][y] = __expf(c_frag[x][y] - row_m1);
                row_l1 += c_frag[x][y];
                c_frag[x][y + 4] = __expf(c_frag[x][y + 4] - row_m2);
                row_l2 += c_frag[x][y + 4];
            }
        }

        #pragma unroll
        for (int x = 3; x >= 1; x /= 2){
            float row_l_other = __shfl_xor_sync(0xffffffff, row_l1, x, 4);
            row_l1 += row_l_other;
            row_l_other = __shfl_xor_sync(0xffffffff, row_l2, x, 4);
            row_l2 += row_l_other;
        }

        float row_m_new1 = fmaxf(row_m1, row_m_prev1);
        float row_m_new2 = fmaxf(row_m2, row_m_prev2);
        float row_l_new1 = (__expf(row_m_prev1 - row_m_new1) * row_l_prev1) + (__expf(row_m1 - row_m_new1) * row_l1);
        float row_l_new2 = (__expf(row_m_prev2 - row_m_new2) * row_l_prev2) + (__expf(row_m2 - row_m_new2) * row_l2);

        #pragma unroll
        for (int x = 0; x < param.Bc / 16; x++){
            #pragma unroll
            for (int y = 0; y < 4; y++){
                d_frag[x][y] = pack_float_to_uint32(c_frag[x][2*y], c_frag[x][2*y+1]);
            }
        }

        __syncthreads();

        float factor1 = 1 / row_l_new1;
        float factor2 = row_l_prev1 * __expf(row_m_prev1 - row_m_new1);
        float factor3 = __expf(row_m1 - row_m_new1);

        float factor4 = 1 / row_l_new2;
        float factor5 = row_l_prev2 * __expf(row_m_prev2 - row_m_new2);
        float factor6 = __expf(row_m2 - row_m_new2);

        // S = S * V
        #pragma unroll
        for (int x = 0; x < param.d / 16; x++){
            memset(c_frag, 0, sizeof(c_frag));
            #pragma unroll
            for(int y = 0; y < param.Bc / 16; y++){
                uint32_t bOffsetPtr = __cvta_generic_to_shared(&Vj[y*param.d*16+x*16+(lane_id%16)*param.d]);
                LDMATRIX_X2_T(b_frag[0], b_frag[1], bOffsetPtr);
                bOffsetPtr = __cvta_generic_to_shared(&Vj[y*param.d*16+x*16+(lane_id%16)*param.d+8]);;
                LDMATRIX_X2_T(b_frag[2], b_frag[3], bOffsetPtr);

                HMMA16816F32(c_frag[0][0], c_frag[0][1], c_frag[0][4], c_frag[0][5], \
                             d_frag[y][0], d_frag[y][2], d_frag[y][1], d_frag[y][3], \
                             b_frag[0], b_frag[1], \
                             c_frag[0][0], c_frag[0][1], c_frag[0][4], c_frag[0][5]);

                HMMA16816F32(c_frag[0][2], c_frag[0][3], c_frag[0][6], c_frag[0][7], \
                             d_frag[y][0], d_frag[y][2], d_frag[y][1], d_frag[y][3], \
                             b_frag[2], b_frag[3], \
                             c_frag[0][2], c_frag[0][3], c_frag[0][6], c_frag[0][7]);

                __syncthreads();
            }

            int offset = warp_id * param.d * 16 + x * 16 + (lane_id / 4) * param.d + (lane_id % 4) * 2;

            O[offset]     = factor1 * ((factor2 * O[offset    ]) + (factor3 * c_frag[0][0]));
            O[offset + 1] = factor1 * ((factor2 * O[offset + 1]) + (factor3 * c_frag[0][1]));
            O[offset + 8] = factor1 * ((factor2 * O[offset + 8]) + (factor3 * c_frag[0][2]));
            O[offset + 9] = factor1 * ((factor2 * O[offset + 9]) + (factor3 * c_frag[0][3]));

            offset += 8 * param.d;

            O[offset]     = factor4 * ((factor5 * O[offset    ]) + (factor6 * c_frag[0][4]));
            O[offset + 1] = factor4 * ((factor5 * O[offset + 1]) + (factor6 * c_frag[0][5]));
            O[offset + 8] = factor4 * ((factor5 * O[offset + 8]) + (factor6 * c_frag[0][6]));
            O[offset + 9] = factor4 * ((factor5 * O[offset + 9]) + (factor6 * c_frag[0][7]));
        }

        __syncthreads();

        row_l_prev1 = row_l_new1;
        row_l_prev2 = row_l_new2;
        row_m_prev1 = row_m_new1;
        row_m_prev2 = row_m_new2;
    }
}