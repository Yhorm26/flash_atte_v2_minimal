#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <random>
#include <sys/time.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include "flash.h"


#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

std::default_random_engine generator(18);
std::uniform_real_distribution<float> distribution(0.0f, 10.0f);


void verfiy(
    float* O, 
    float* O_host,
    float range_of_error)
{
    int error=0;
    printf("===================start verfiy===================\n");
    for(int i=0;i<BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD;i++)
    {
        float device_out = O_host[i];
        if((fabs(O_host[i] - O[i]))/O_host[i] > range_of_error || std::isnan(device_out) || std::isinf(device_out))
        {
            printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, O_host[i], O[i]);
            error++;
            break;
        }        
    }
    printf("==================finish,error:%d==================\n",error);
}


void attention_forward_cpu(
    float* Q, 
    float* K, 
    float* V, 
    float sqrt_head_dim, 
    float* output)
{

    const int head_size = SEQ_LEN * HEAD_EMBD;
    
    // 临时存储注意力分数
    float* scores = new float[SEQ_LEN * SEQ_LEN];

    for (int b = 0; b < BATCH_SIZE; ++b) {
        
        for (int h = 0; h < N_HEAD; ++h) {
            // 获取当前head的指针偏移量
            const int base_offset = b * N_HEAD * head_size + h * head_size;
            const float* Q_ptr = Q + base_offset;
            const float* K_ptr = K + base_offset;
            const float* V_ptr = V + base_offset;
            float* out_ptr = output + base_offset;

            // 1. 手动实现QK^T矩阵乘法
            for (int i = 0; i < SEQ_LEN; ++i) {
                for (int j = 0; j < SEQ_LEN; ++j) {
                    float sum = 0.0f;
                    for (int k = 0; k < HEAD_EMBD; ++k) {
                        sum += Q_ptr[i * HEAD_EMBD + k] * K_ptr[j * HEAD_EMBD + k];
                    }
                    scores[i * SEQ_LEN + j] = sum * sqrt_head_dim;
                }
            }

            // 2. Softmax计算
            for (int i = 0; i < SEQ_LEN; ++i) {
                float max_val = -INFINITY;
                float* row = scores + i * SEQ_LEN;
                
                // 计算行最大值
                for (int j = 0; j < SEQ_LEN; ++j) {
                    if (row[j] > max_val) max_val = row[j];
                }

                // 计算指数和
                float sum = 0.0f;
                for (int j = 0; j < SEQ_LEN; ++j) {
                    row[j] = expf(row[j] - max_val);
                    sum += row[j];
                }

                // 归一化
                for (int j = 0; j < SEQ_LEN; ++j) {
                    row[j] /= sum;
                }
            }

            // 4. 手动实现注意力加权矩阵乘法
            for (int i = 0; i < SEQ_LEN; ++i) {
                for (int k = 0; k < HEAD_EMBD; ++k) {
                    float sum = 0.0f;
                    for (int j = 0; j < SEQ_LEN; ++j) {
                        sum += scores[i * SEQ_LEN + j] * V_ptr[j * HEAD_EMBD + k];
                    }
                    out_ptr[i * HEAD_EMBD + k] = sum;
                }
            }
        }
    }

    delete[] scores;
}


void launchKernel(
    mykernelParamType param, 
    void (*kernel)(mykernelParamType), 
    int grid_x, int grid_y, int grid_z, 
    int block_x, 
    int sram_size, 
    float* O,
    float* O_host,
    float* O_device,
    float range_of_error) 
{
    dim3 grid_dim(grid_x, grid_y, grid_z);
    dim3 block_dim(block_x);

    // 预热
    kernel<<<grid_dim, block_dim, sram_size>>>(param);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(O_host, O_device, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float), hipMemcpyDeviceToHost);

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    float time_elapsed=0.0;

    for (int i = 0; i < 100; i++){
        kernel<<<grid_dim, block_dim, sram_size>>>(param);
    }
    
    hipEventRecord(stop,0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed,start,stop);

    printf("kernel time: %f us\n", time_elapsed*1000 / 100);
    printf("Verify the result of kernel function\n");

    verfiy(O, O_host, range_of_error);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void launchKernel(
    mykernelParamType2 param, 
    void (*kernel)(mykernelParamType2), 
    int grid_x, int grid_y, int grid_z, 
    int block_x, 
    int sram_size, 
    float* O,
    float* O_host,
    float* O_device,
    float range_of_error) 
{
    dim3 grid_dim(grid_x, grid_y, grid_z);
    dim3 block_dim(block_x);

    // 预热
    kernel<<<grid_dim, block_dim, sram_size>>>(param);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(O_host, O_device, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float), hipMemcpyDeviceToHost);

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    float time_elapsed=0.0;

    for (int i = 0; i < 100; i++){
        kernel<<<grid_dim, block_dim, sram_size>>>(param);
    }
    
    hipEventRecord(stop,0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed,start,stop);

    printf("kernel time: %f us\n", time_elapsed*1000 / 100);
    printf("Verify the result of kernel function\n");

    verfiy(O, O_host, range_of_error);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


int main(){
    float *Q      = (float*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));
    float *K      = (float*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));
    float *V      = (float*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));
    float *O      = (float*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));
    float *O_host = (float*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));

    half *Q_half = (half*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half));
    half *K_half = (half*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half));
    half *V_half = (half*)malloc(BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half));

    float *Q_device,*K_device,*V_device, *O_device;
    hipMalloc((void**)&Q_device, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));
    hipMalloc((void**)&K_device, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));
    hipMalloc((void**)&V_device, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));
    hipMalloc((void**)&O_device, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float));

    half *Q_device_half,*K_device_half,*V_device_half;
    hipMalloc((void**)&Q_device_half, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half));
    hipMalloc((void**)&K_device_half, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half));
    hipMalloc((void**)&V_device_half, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half));

    for(int i = 0; i < BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD; i++)
    {
        Q[i] = distribution(generator);
        K[i] = distribution(generator);
        V[i] = distribution(generator);
        O[i] = 0.0f;

        Q_half[i] = __float2half(Q[i]);
        K_half[i] = __float2half(K[i]);
        V_half[i] = __float2half(V[i]);
    }
    
    hipMemcpy(Q_device, Q, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(K_device, K, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(V_device, V, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(float),hipMemcpyHostToDevice);

    hipMemcpy(Q_device_half, Q_half, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(K_device_half, K_half, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(V_device_half, V_half, BATCH_SIZE*N_HEAD*SEQ_LEN*HEAD_EMBD*sizeof(half),hipMemcpyHostToDevice);

    mykernelParamType param;
    param.Q             = Q_device;
    param.K             = K_device;
    param.V             = V_device;
    param.O             = O_device;
    param.N             = SEQ_LEN;
    param.d             = HEAD_EMBD;
    param.Bc            = Bc1;
    param.Br            = Br1;
    param.Tc            = ceil(SEQ_LEN / param.Bc);
    param.Tr            = ceil(SEQ_LEN / param.Br);
    param.softmax_scale = 1.0 / sqrt(HEAD_EMBD);

    // 计算每个线程块所需的SRAM大小
    int sram_size = ((2 * param.Bc + param.Br) * HEAD_EMBD * sizeof(float)) + (param.Bc * param.Br * sizeof(float));
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
    attention_forward_cpu(Q, K, V, param.softmax_scale, O);

    // ************************************kernel_1***************************************************************************
    launchKernel(param,forward_kernel_1, param.Tr, N_HEAD, BATCH_SIZE, param.Br, sram_size, O, O_host, O_device, 0.0001);
    // ************************************kernel_2***************************************************************************
    launchKernel(param,forward_kernel_2, param.Tr, N_HEAD, BATCH_SIZE, param.Br * 8, sram_size, O, O_host, O_device, 0.0001);
    // ************************************kernel_3***************************************************************************
    mykernelParamType2 param2;
    param2.Q             = Q_device_half;
    param2.K             = K_device_half;
    param2.V             = V_device_half;
    param2.O             = O_device;
    param2.N             = SEQ_LEN;
    param2.d             = HEAD_EMBD;
    param2.Bc            = Bc2;
    param2.Br            = Br2;
    param2.Tc            = ceil(SEQ_LEN / param2.Bc);
    param2.Tr            = ceil(SEQ_LEN / param2.Br);
    param2.softmax_scale = 1.0 / sqrt(HEAD_EMBD);

    int sram_size2 = ((2 * param2.Bc + param2.Br) * HEAD_EMBD * sizeof(half)) + param2.Bc * param2.Br * sizeof(half);
    printf("Max shared memory: %d, kernel_3 requested shared memory: %d \n", max_sram_size, sram_size + param.d * param.Br * 4);

    launchKernel(param2, forward_kernel_3, param2.Tr, N_HEAD, BATCH_SIZE, (param2.Br/16)*(param2.Bc/16)*32, sram_size2, O, O_host, O_device, 0.04);
    // ****************************************************************************************************************************

    hipFree(Q_device);
    hipFree(K_device);
    hipFree(V_device);
    hipFree(O_device);
    hipFree(Q_device_half);
    hipFree(K_device_half);
    hipFree(V_device_half);
    
    free(Q);
    free(K);
    free(V);
    free(O);
    free(O_host);
    free(Q_half);
    free(K_half);
    free(V_half);
    
    return 0;
}